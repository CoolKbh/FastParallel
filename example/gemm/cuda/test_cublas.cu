#include <sys/time.h>
#include <hip/hip_runtime_api.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include <stdio.h>

int main () {
    int m = 4096, n = 4096, k = 4096;
    printf("shape: (%d %d) x (%d %d)\n", m ,k, k, n);
    int start_algo = HIPBLAS_GEMM_DEFAULT;
    
}