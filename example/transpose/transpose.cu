#include "hip/hip_runtime.h"
#include "cuda_gemm.h"

int upScale(int x, int y) {
    return (x + y - 1) / y;
}

__global__ void gemm_native(float *a, float *b, float *c, int N, int M, int K){
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = blockIdx.y * blockDim.y + threadIdx.y;

    if(tx >= M || ty >= N)
        return;

    int c_temp = 0;
    for(int i = 0; i < K; i++) {
        // c[tx * N + ty] += a[tx * K + i] * b[i * N + ty];
        c_temp += a[ty * K + i] * b[i * N + tx];  //结果一致
    }
    c[ty * N + tx] = c_temp;
}

__global__ void gemm_kernel_1(float *a, float *b, float *c, int N, int M, int K){
    int tx = blockIdx.x * blockDim.x + (threadIdx.x / 32);
    int ty = blockIdx.y * blockDim.x + (threadIdx.x % 32);

    if(tx >= M || ty >= N)
        return;

    int c_temp = 0;
    for(int i = 0; i < K; i++) {
        // c[tx * N + ty] += a[tx * K + i] * b[i * N + ty];
        c_temp += a[ty * K + i] * b[i * N + tx];  //结果一致
    }
    c[ty * N + tx] = c_temp;
}

// shared memory 
__global__ void gemm_kernel_2(float *a, float *b, float *c, int N, int M, int K){
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = blockIdx.y * blockDim.y + threadIdx.y;

    // if(tx >= N || ty >= M)
    //     return;

    // shared memory
    __shared__ float As[32][32];
    __shared__ float Bs[32][32];

    int Ks = 32;
    float c_temp = 0;

    for(int i = 0; i < K / Ks; ++i) {
        As[threadIdx.y][threadIdx.x] = a[ty * K + i * Ks + threadIdx.x];
        Bs[threadIdx.y][threadIdx.x] = b[i * Ks * N + tx + threadIdx.y * N];
        __syncthreads();

        for(int j = 0; j < Ks; ++j) {
            c_temp += As[threadIdx.y][j] * Bs[j][threadIdx.x]; 
        }
        __syncthreads();  
    }
    c[ty * N + tx] = c_temp;
}

inline void run_gemm_native(float *a, float *b, float *c, int N, int M, int K) {
    dim3 dimGrid(upScale(M, 32), upScale(N, 32));
    dim3 dimBlock(32, 32);

    gemm_native<<<dimGrid, dimBlock>>>(a, b, c, N, M, K);
}

void gemm(float *a, float *b, float *c, int N, int M, int K){
    const int BLOCK_SIZE_M = 128;
    const int BLOCK_SIZE_K = 8;
    const int BLOCK_SIZE_N = 128;
    const int THREAD_SIZE_Y = 32;
    const int THREAD_SIZE_X = 32;

    dim3 dimBlock(THREAD_SIZE_X, THREAD_SIZE_Y);
    dim3 dimGrid(upScale(M, THREAD_SIZE_X), upScale(N, THREAD_SIZE_Y));
    // dim3 dimBlock(2, 2);
    // dim3 dimGrid(4, 4);

    int niter = 1000;
    float msecTotal = 0;
    
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventRecord(start));

    int kernel_num = 0;

    for(int i = 0; i < niter; i++) {
       gemm_kernel_2<<<dimGrid, dimBlock>>>(a, b, c, N, M, K); 
    }

    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

    double msecPerMatrixMul[2] = {0, 0};
    double gigaFlops[2] = {0, 0};
    double flopsPerMatrixMul = 2.0 * M * N * K;

    msecPerMatrixMul[0] = msecTotal / niter;
    gigaFlops[0] = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul[0] / 1000.0f);
    printf("My gemm Performance = %.2f GFlop/s, Time = %.3f msec, Size = %.0f Ops\n", 
            gigaFlops[0],
            msecPerMatrixMul[0],
            flopsPerMatrixMul);
    
}